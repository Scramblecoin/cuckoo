#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"

#ifndef MAXSOLS
#define MAXSOLS 1
#endif
#define MAXINT (1<<31-1)

#if SIZESHIFT <= 32
  typedef u32 nonce_t;
  typedef u32 node_t;
  typedef uint2 edge_t;
#define make_edge make_uint2
#else
  typedef u64 nonce_t;
  typedef u64 node_t;
  typedef ulong2 edge_t;
#define make_edge make_ulong2
#endif
typedef unsigned long long ull;

static __device__ __forceinline__ bool operator== (edge_t a, edge_t b) { return a.x == b.x && a.y == b.y; }

// d(evice s)ipnode
#if (__CUDA_ARCH__  >= 320) // redefine ROTL to use funnel shifter, 3% speed gain

static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
static __device__ __forceinline__ void operator^= (uint2 &a, uint2 b) { a.x ^= b.x, a.y ^= b.y; }
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) {
  asm("{\n\tadd.cc.u32 %0,%2,%4;\n\taddc.u32 %1,%3,%5;\n\t}\n\t"
    : "=r"(a.x), "=r"(a.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
}
#undef ROTL
__inline__ __device__ uint2 ROTL(const uint2 a, const int offset) {
  uint2 result;
  if (offset >= 32) {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t" : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}
__device__ node_t dipnode(siphash_ctx &ctx, nonce_t nce, u32 uorv) {
  uint2 nonce = vectorize(2*nce + uorv);
  uint2 v0 = ctx.v2[0], v1 = ctx.v2[1], v2 = ctx.v2[2], v3 = ctx.v2[3] ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= vectorize(0xff);
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return devectorize(v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#else

__device__ node_t dipnode(siphash_ctx &ctx, nonce_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = ctx.v[0], v1 = ctx.v[1], v2 = ctx.v[2], v3 = ctx.v[3] ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#endif

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (SIZE >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// SIZE >> IDXSHIFT == HALFSIZE >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif
// grow with cube root of size, hardly affected by trimming
#ifndef MAXPATHLEN
#define MAXPATHLEN (8 << (SIZESHIFT/3))
#endif

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset {
public:
  u64 *bits;
  __device__ void reset(nonce_t n) {
    bits[n/64] |= 1LL << (n%64);
  }
  __device__ bool test(node_t n) const {
    return !((bits[n/64] >> (n%64)) & 1);
  }
  __device__ u64 block(node_t n) const {
    return ~bits[n/64];
  }
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (HALFSIZE >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set {
public:
  u32 *bits;
  __device__ void reset() {
    memset(bits, 0, TWICE_WORDS * sizeof(u32));
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

#define CUCKOO_SIZE (SIZE >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by SIZESHIFT
#define KEYBITS (64-SIZESHIFT)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash {
public:
  u64 *cuckoo;
  u32 nset;

  void set(node_t u, node_t oldv, node_t newv) {
    u64 niew = (u64)u << SIZESHIFT | newv;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 old = cuckoo[ui];
      if (old == 0 || (old >> SIZESHIFT) == (u & KEYMASK)) {
        cuckoo[ui] = niew;
        return;
      }
    }
  }
  __device__ bool dset(node_t u, node_t oldv, node_t newv) {
    u64 old, exp = (oldv ? (u64)u << SIZESHIFT | oldv : 0), nuw = (u64)u << SIZESHIFT | newv;
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      old = atomicCAS((ull *)&cuckoo[ui], (ull)exp, (ull)nuw);
      if (old == exp) {
        return true;
      }
      if ((old >> SIZESHIFT) == (u & KEYMASK)) {
        return false;
      }
    }
  }
  node_t operator[](node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> SIZESHIFT) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & (SIZE-1));
      }
    }
  }
  __device__ node_t node(node_t u) const {
    for (node_t ui = u >> IDXSHIFT; ; ui = (ui+1) & CUCKOO_MASK) {
      u64 cu = cuckoo[ui];
      if (!cu)
        return 0;
      if ((cu >> SIZESHIFT) == (u & KEYMASK)) {
        assert(((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
        return (node_t)(cu & (SIZE-1));
      }
    }
  }
};

struct noncedge_t {
  nonce_t nonce;
  edge_t edge;
};

class cuckoo_ctx {
public:
  siphash_ctx sip_ctx;
  shrinkingset alive;
  twice_set nonleaf;
  cuckoo_hash cuckoo;
  noncedge_t sols[MAXSOLS][PROOFSIZE];
  u32 nsols;
  nonce_t gpu_nonce_lim;
  u32 nthreads;

  cuckoo_ctx(const char* header, nonce_t gpulim, u32 n_threads) {
    setheader(&sip_ctx, header);
    gpu_nonce_lim = gpulim & ~0x3f; // need multiple of 64
    nthreads = n_threads;
    nsols = 0;
  }
};

__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_ctx sip_ctx = ctx->sip_ctx; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (nonce_t block = id*64; block < HALFSIZE; block += ctx->nthreads*64) {
    u64 alive64 = alive.block(block);
    for (nonce_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive64);
      nonce += ffs; alive64 >>= ffs;
      node_t u = dipnode(sip_ctx, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}

__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_ctx sip_ctx = ctx->sip_ctx;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (nonce_t block = id*64; block < HALFSIZE; block += ctx->nthreads*64) {
    u64 alive64 = alive.block(block);
    for (nonce_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive64);
      nonce += ffs; alive64 >>= ffs;
      node_t u = dipnode(sip_ctx, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}

__device__ u32 dpath(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo.node(u)) {
    if (nu++ >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      return ~0;
    }
    us[nu] = u;
    if (nu>=2 && u==us[nu-2])
      return ~0;
  }
  us[nu+1] = 0;
  return nu;
}

__global__ void find_cycles(cuckoo_ctx *ctx) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  node_t us[MAXPATHLEN+2], vs[MAXPATHLEN+2];
  shrinkingset &alive = ctx->alive;
  siphash_ctx sip_ctx = ctx->sip_ctx;
  cuckoo_hash &cuckoo = ctx->cuckoo;
  for (nonce_t block = id*64; block < ctx->gpu_nonce_lim; block += ctx->nthreads*64) {
    u64 alive64 = alive.block(block);
    for (nonce_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive64);
      nonce += ffs; alive64 >>= ffs;
      node_t u0 = dipnode(sip_ctx, nonce, 0)<<1, v0 = dipnode(sip_ctx, nonce, 1)<<1|1;
      if (u0 == 0) // ignore vertex 0 so it can be used as nil for cuckoo[]
        continue;
      us[0] = u0; vs[0] = v0;
      int nredo = 0;
redo: if (nredo++) printf("redo\n");
      node_t u1 = cuckoo.node(u0), v1 = cuckoo.node(v0);

      u32 nu, nv;
      nonce_t u=u0;
      for (nu = 0; u; u = cuckoo.node(u)) {
        if (nu++ >= MAXPATHLEN) {
          while (nu-- && us[nu] != u) ;
          if (nu == ~0)
            printf("maximum path length exceeded\n");
          else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
          break;
        }
        us[nu] = u;
        if (nu>=2 && u==us[nu-2])
          break;
      }
      if (u) {
       //printf("oops\n");
       continue;
      }
      us[nu+1] = 0;

      nonce_t v=v0;
      for (nv = 0; v; v = cuckoo.node(v)) {
        if (nv++ >= MAXPATHLEN) {
          while (nv-- && vs[nv] != v) ;
          if (nv == ~0)
            printf("maximum path length exceeded\n");
          else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
          break;
        }
        vs[nv] = v;
        if (nv>=2 && v==vs[nv-2])
          break;
      }
      if (v) {
       //printf("oops\n");
       continue;
      }
      vs[nv+1] = 0;

      // u32 nu = dpath(cuckoo, u1, us), nv = dpath(cuckoo, v1, vs);

      if (nu==~0 || nv==~0) continue;
      if (us[nu] == vs[nv]) {
        u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        u32 len = nu + nv + 1;
        printf("% 4d-cycle found at %d:%d%%\n", len, id, (u32)(nonce*100L/HALFSIZE));
        if (len == PROOFSIZE) {
          u32 slot = atomicInc(&ctx->nsols, MAXINT);
          if (slot < MAXSOLS) {
            noncedge_t *ne = &ctx->sols[slot][0];
            ne++->edge = make_edge(*us, *vs);
            while (nu--)
              ne++->edge = make_edge(us[(nu + 1)&~1], us[nu | 1]); // u's in even position; v's in odd
            while (nv--)
              ne++->edge = make_edge(vs[nv | 1], vs[(nv + 1)&~1]); // u's in odd position; v's in even
          }
        }
        continue;
      }
      if (nu < nv) {
        while (nu--)
          if (!cuckoo.dset(us[nu+1], us[nu+2], us[nu])) goto redo;
        if (!cuckoo.dset(u0, u1, v0)) goto redo;
      } else {
        while (nv--)
          if (!cuckoo.dset(vs[nv+1], vs[nv+2], vs[nv])) goto redo;
        if (!cuckoo.dset(v0, v1, u0)) goto redo;
      }
    }
  }
}

u32 path(cuckoo_hash &cuckoo, node_t u, node_t *us) {
  u32 nu;
  for (nu = 0; u; u = cuckoo[u]) {
    if (nu++ >= MAXPATHLEN) {
      while (nu-- && us[nu] != u) ;
      if (nu == ~0)
        printf("maximum path length exceeded\n");
      else printf("illegal % 4d-cycle\n", MAXPATHLEN-nu);
      return ~0;
    }
    us[nu] = u;
    if (nu>=2 && u==us[nu-2])
      return ~0;
  }
  us[nu+1] = 0;
  return nu;
}

void find_more_cycles(cuckoo_ctx *ctx, cuckoo_hash &cuckoo, u64 *bits) {
  node_t us[MAXPATHLEN+2], vs[MAXPATHLEN+2];
  for (nonce_t block = ctx->gpu_nonce_lim; block < HALFSIZE; block += 64) {
    u64 alive64 = ~bits[block/64];
    for (nonce_t nonce = block-1; alive64; ) { // -1 compensates for 1-based ffs
      // printf("nonce %d\n", nonce);
      u32 ffs = __builtin_ffsll(alive64);
      nonce += ffs; alive64 >>= ffs;
      node_t u0=sipnode(&ctx->sip_ctx, nonce, 0), v0=sipnode(&ctx->sip_ctx, nonce, 1);
      if (u0 == 0) // ignore vertex 0 so it can be used as nil for cuckoo[]
        continue;
      us[0] = u0; vs[0] = v0;
      node_t u1 = cuckoo[u0], v1 = cuckoo[v0];
      u32 nu = path(cuckoo, u1, us), nv = path(cuckoo, v1, vs);
      if (nu==~0 || nv==~0) continue;
      if (us[nu] == vs[nv]) {
        u32 min = nu < nv ? nu : nv;
        for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
        u32 len = nu + nv + 1;
        printf("% 4d-cycle found at 0:%d%%\n", len, (u32)(nonce*100L/HALFSIZE));
        if (len == PROOFSIZE) {
          u32 slot = ctx->nsols++;
          if (slot < MAXSOLS) {
            noncedge_t *ne = &ctx->sols[slot][0];
            ne++->edge = make_edge(*us, *vs);
            while (nu--)
              ne++->edge = make_edge(us[(nu + 1)&~1], us[nu | 1]); // u's in even position; v's in odd
            while (nv--)
              ne++->edge = make_edge(vs[nv | 1], vs[(nv + 1)&~1]); // u's in odd position; v's in even
          }
        }
        continue;
      }
      if (nu < nv) {
        while (nu--)
          cuckoo.set(us[nu+1], us[nu+2], us[nu]);
        cuckoo.set(u0, u1, v0);
      } else {
        while (nv--)
          cuckoo.set(vs[nv+1], vs[nv+2], vs[nv]);
        cuckoo.set(v0, v1, u0);
      }
      if (ffs & 64) break; // can't shift by 64
    }
  }
}

__global__ void find_nonces(cuckoo_ctx *ctx) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  shrinkingset &alive = ctx->alive;
  siphash_ctx sip_ctx = ctx->sip_ctx;

  for (nonce_t block = id * 64; block < HALFSIZE; block += ctx->nthreads * 64) {
    u64 alive64 = alive.block(block);
    for (nonce_t nonce = block - 1; alive64;) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive64);
      nonce += ffs; alive64 >>= ffs;
      edge_t edge = make_edge(dipnode(sip_ctx,nonce,0)<<1, dipnode(sip_ctx,nonce,1)<<1|1);
      for (u32 i = 0; i < ctx->nsols; i++) {
        noncedge_t *sol = ctx->sols[i];
        for (u32 j = 0; j < PROOFSIZE; j++) {
          if (sol[j].edge == edge)
            sol[j].nonce = nonce;
        }
      }
    }
  }
}

int noncedge_cmp(const void *a, const void *b) {
  return ((noncedge_t *)a)->nonce - ((noncedge_t *)b)->nonce;
}

#include <unistd.h>

int main(int argc, char **argv) {
  int gpu_pct = 50;
  int nthreads = 1;
  int ntrims   = 1 + (PART_BITS+3)*(PART_BITS+4)/2;
  int tpb = 0;
  const char *header = "";
  int c;
  while ((c = getopt (argc, argv, "h:m:n:g:t:p:")) != -1) {
    switch (c) {
      case 'h':
        header = optarg;
        break;
      case 'n':
        ntrims = atoi(optarg);
        break;
      case 'g':
        gpu_pct = atoi(optarg);
        break;
      case 't':
        nthreads = atoi(optarg);
        break;
      case 'p':
        tpb = atoi(optarg);
        break;
    }
  }
  if (!tpb) // if not set, then default threads per block to roughly square root of threads
    for (tpb = 1; tpb*tpb < nthreads; tpb *= 2) ;

  printf("Looking for %d-cycle on cuckoo%d(\"%s\") with 50%% edges, %d trims, %d%% gpu, %d threads %d per block\n",
               PROOFSIZE, SIZESHIFT, header, ntrims, gpu_pct, nthreads, tpb);
  u64 edgeBytes = HALFSIZE/8, nodeBytes = TWICE_WORDS*sizeof(u32);

  nonce_t gpu_lim = HALFSIZE*gpu_pct/100 & ~0x3f;
  cuckoo_ctx ctx(header, gpu_lim, nthreads);
  checkCudaErrors(hipMalloc((void**)&ctx.alive.bits, edgeBytes));
  checkCudaErrors(hipMemset(ctx.alive.bits, 0, edgeBytes));
  checkCudaErrors(hipMalloc((void**)&ctx.nonleaf.bits, nodeBytes));

  int edgeUnit=0, nodeUnit=0;
  u64 eb = edgeBytes, nb = nodeBytes;
  for (; eb >= 1024; eb>>=10) edgeUnit++;
  for (; nb >= 1024; nb>>=10) nodeUnit++;
  printf("Using %d%cB edge and %d%cB node memory.\n",
     (int)eb, " KMGT"[edgeUnit], (int)nb, " KMGT"[nodeUnit]);

  cuckoo_ctx *device_ctx;
  checkCudaErrors(hipMalloc((void**)&device_ctx, sizeof(cuckoo_ctx)));
  hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);

  for (u32 round=0; round < ntrims; round++) {
    for (u32 uorv = 0; uorv < 2; uorv++) {
      for (u32 part = 0; part <= PART_MASK; part++) {
        checkCudaErrors(hipMemset(ctx.nonleaf.bits, 0, nodeBytes));
        count_node_deg<<<nthreads/tpb,tpb>>>(device_ctx, uorv, part);
        kill_leaf_edges<<<nthreads/tpb,tpb>>>(device_ctx, uorv, part);
      }
    }
  }

  u64 *bits;
  bits = (u64 *)calloc(HALFSIZE/64, sizeof(u64));
  assert(bits != 0);
  hipMemcpy(bits, ctx.alive.bits, (HALFSIZE/64) * sizeof(u64), hipMemcpyDeviceToHost);

  u64 cnt = 0;
  for (int i = 0; i < HALFSIZE/64; i++)
    cnt += __builtin_popcountll(~bits[i]);
  u32 load = (u32)(100 * cnt / CUCKOO_SIZE);
  printf("final load %d%%\n", load);

  if (load >= 90) {
    printf("overloaded! exiting...");
    exit(0);
  }

  checkCudaErrors(hipFree(ctx.nonleaf.bits));
  u32 cuckooBytes = CUCKOO_SIZE * sizeof(u64);
  checkCudaErrors(hipMalloc((void**)&ctx.cuckoo.cuckoo, cuckooBytes));
  checkCudaErrors(hipMemset(ctx.cuckoo.cuckoo, 0, cuckooBytes));
  
  hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);
  find_cycles<<<nthreads/tpb,tpb>>>(device_ctx);
  hipMemcpy(&ctx, device_ctx, sizeof(cuckoo_ctx), hipMemcpyDeviceToHost);

  cuckoo_hash *cuckoo = new cuckoo_hash();
  cuckoo->cuckoo = (u64 *)calloc(CUCKOO_SIZE, sizeof(u64));
  assert(cuckoo->cuckoo != 0);
  hipMemcpy(cuckoo->cuckoo, ctx.cuckoo.cuckoo, cuckooBytes, hipMemcpyDeviceToHost);

  cnt = 0;
  for (int i = 0; i < CUCKOO_SIZE; i++)
    cnt += (cuckoo->cuckoo[i] != 0);
  printf("%lu gpu edges\n", cnt);

  find_more_cycles(&ctx, *cuckoo, bits);
  free(cuckoo->cuckoo);

  if (ctx.nsols) {
    hipMemcpy(device_ctx, &ctx, sizeof(cuckoo_ctx), hipMemcpyHostToDevice);
    find_nonces<<<nthreads/tpb, tpb>>>(device_ctx);
    hipMemcpy(&ctx, device_ctx, sizeof(cuckoo_ctx), hipMemcpyDeviceToHost);

    for (u32 i = 0; i < ctx.nsols; i++) {
      printf("Solution");
      qsort(ctx.sols[i], PROOFSIZE, sizeof(noncedge_t), noncedge_cmp);
      for (u32 j = 0; j < PROOFSIZE; j++)
        printf(" %jx", (uintmax_t)ctx.sols[i][j].nonce);
      printf("\n");
    }
  }

  checkCudaErrors(hipFree(ctx.cuckoo.cuckoo));
  checkCudaErrors(hipFree(ctx.alive.bits));
  return 0;
}
