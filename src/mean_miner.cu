#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include "cuckoo.h"
#include <sys/time.h>

#include "cuckoo_miner/cuda_miner_adds.h"

// d(evice s)ipnode
#if (__CUDA_ARCH__  >= 320) // redefine ROTL to use funnel shifter, 3% speed gain

static __device__ __forceinline__ uint2 operator^ (uint2 a, uint2 b) { return make_uint2(a.x ^ b.x, a.y ^ b.y); }
static __device__ __forceinline__ void operator^= (uint2 &a, uint2 b) { a.x ^= b.x, a.y ^= b.y; }
static __device__ __forceinline__ void operator+= (uint2 &a, uint2 b) {
  asm("{\n\tadd.cc.u32 %0,%2,%4;\n\taddc.u32 %1,%3,%5;\n\t}\n\t"
    : "=r"(a.x), "=r"(a.y) : "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
}
#undef ROTL
__inline__ __device__ uint2 ROTL(const uint2 a, const int offset) {
  uint2 result;
  if (offset >= 32) {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.x), "r"(a.y), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.y), "r"(a.x), "r"(offset));
  } else {
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.x) : "r"(a.y), "r"(a.x), "r"(offset));
    asm("shf.l.wrap.b32 %0, %1, %2, %3;" : "=r"(result.y) : "r"(a.x), "r"(a.y), "r"(offset));
  }
  return result;
}
__device__ __forceinline__ uint2 vectorize(const uint64_t x) {
  uint2 result;
  asm("mov.b64 {%0,%1},%2; \n\t" : "=r"(result.x), "=r"(result.y) : "l"(x));
  return result;
}
__device__ __forceinline__ uint64_t devectorize(uint2 x) {
  uint64_t result;
  asm("mov.b64 %0,{%1,%2}; \n\t" : "=l"(result) : "r"(x.x), "r"(x.y));
  return result;
}
__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  uint2 nonce = vectorize(2*nce + uorv);
  uint2 v0 = vectorize(keys.k0), v1 = vectorize(keys.k1), v2 = vectorize(keys.k2), v3 = vectorize(keys.k3) ^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= vectorize(0xff);
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return devectorize(v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#else

__device__ node_t dipnode(siphash_keys &keys, edge_t nce, u32 uorv) {
  u64 nonce = 2*nce + uorv;
  u64 v0 = keys.k0, v1 = keys.k0, v2 = keys.k2, v3 = keys.k3^ nonce;
  SIPROUND; SIPROUND;
  v0 ^= nonce;
  v2 ^= 0xff;
  SIPROUND; SIPROUND; SIPROUND; SIPROUND;
  return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

#endif

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <vector>
#include <bitset>

// algorithm/performance parameters

// EDGEBITS/NEDGES/EDGEMASK defined in cuckoo.h

// The node bits are logically split into 3 groups:
// XBITS 'X' bits (most significant), YBITS 'Y' bits, and ZBITS 'Z' bits (least significant)
// Here we have the default XBITS=YBITS=7, ZBITS=15 summing to EDGEBITS=29
// nodebits   XXXXXXX YYYYYYY ZZZZZZZZZZZZZZZ
// bit%10     8765432 1098765 432109876543210
// bit/10     2222222 2111111 111110000000000

// The matrix solver stores all edges in a matrix of NX * NX buckets,
// where NX=2^XBITS is the number of possible values of the 'X' bits.
// Edge i between nodes ui = siphash24(2*i) and vi = siphash24(2*i+1)
// resides in the bucket at (uiX,viX)
// In each trimming round, either a matrix row or a matrix column (NX buckets)
// is bucket sorted on uY or vY respectively, and then within each bucket
// uZ or vZ values are counted and edges with a count of only one are eliminated,
// while remaining edges are bucket sorted back on vY or uY respectively.
// When sufficiently many edges have been eliminated, a pair of compression
// rounds remap surviving Z values in each X,Y bucket to fit into 15-YBITS bits,
// allowing the remaining rounds to avoid the sorting on Y and directly
// count YZ values in a cache friendly 32KB.

#ifndef XBITS
// 7 seems to give best performance
#define XBITS 7
#endif

#define YBITS XBITS

// size in bytes of a big bucket entry
#ifndef BIGSIZE
#define BIGSIZE 5
#endif

// YZ compression round; must be even
#ifndef COMPRESSROUND
#define COMPRESSROUND 16
#endif

typedef uint8_t u8;
typedef uint16_t u16;

// node bits have two groups of bucketbits (big and small) and a remaining group of degree bits
const static u32 NX        = 1 << XBITS;
const static u32 XMASK     = NX - 1;
const static u32 NY        = 1 << YBITS;
const static u32 YMASK     = NY - 1;
const static u32 XYBITS    = XBITS + YBITS;
const static u32 NXY       = 1 << XYBITS;
const static u32 ZBITS     = EDGEBITS - XYBITS;
const static u32 NZ        = 1 << ZBITS;
const static u32 ZMASK     = NZ - 1;
const static u32 YZBITS    = YBITS + ZBITS;
const static u32 NYZ       = 1 << YZBITS;
const static u32 YZMASK    = NYZ - 1;
const static u32 YZ1BITS   = 15;  // combined Y and compressed Z bits
const static u32 NYZ1      = 1 << YZ1BITS;
const static u32 MAXNZNYZ1 = NYZ1 > NZ ? NYZ1 : NZ;
const static u32 YZ1MASK   = NYZ1 - 1;
const static u32 Z1BITS    = YZ1BITS - YBITS;
const static u32 NZ1       = 1 << Z1BITS;
const static u32 Z1MASK    = NZ1 - 1;
const static u32 YZ2BITS   = 9;  // more compressed YZ bits
const static u32 NYZ2      = 1 << YZ2BITS;
const static u32 YZ2MASK   = NYZ2 - 1;
const static u32 Z2BITS    = YZ2BITS - YBITS;
const static u32 NZ2       = 1 << Z2BITS;
const static u32 Z2MASK    = NZ2 - 1;
const static u32 YZZBITS   = YZBITS + ZBITS;
const static u32 YZZ1BITS  = YZ1BITS + ZBITS;

const static u32 BIGSLOTBITS   = BIGSIZE * 8;
const static u32 NONYZBITS     = BIGSLOTBITS - YZBITS;
const static u32 NNONYZ        = 1 << NONYZBITS;

const static u32 Z2BUCKETSIZE = NYZ2 >> 3;

// for p close to 0, Pr(X>=k) < e^{-n*p*eps^2} where k=n*p*(1+eps)
// see https://en.wikipedia.org/wiki/Binomial_distribution#Tail_bounds
// eps should be at least 1/sqrt(n*p/64)
// to give negligible bad odds of e^-64.

// 1/32 reduces odds of overflowing z bucket on 2^30 nodes to 2^14*e^-32
// (less than 1 in a billion) in theory. not so in practice (fails first at cuda30 -n 1679)
#ifndef BIGEPS
#define BIGEPS 3/64
#endif

const static u32 ZBUCKETSLOTS = NZ + NZ * BIGEPS;
const static u32 ZBUCKETSIZE = ZBUCKETSLOTS * BIGSIZE;
const static u32 TBUCKETSIZE = ZBUCKETSLOTS * BIGSIZE;

template<u32 BUCKETSIZE, u32 NRENAME, u32 NRENAME1>
struct zbucket {
  u32 size;
  const static u32 RENAMESIZE = 2*NRENAME1 + 2*NRENAME;
  union {
    u8 bytes[BUCKETSIZE];
    struct {
      u32 words[BUCKETSIZE/sizeof(u32) - RENAMESIZE];
      u32 renameu1[NRENAME1];
      u32 renamev1[NRENAME1];
      u32 renameu[NRENAME];
      u32 renamev[NRENAME];
    };
  };
  __device__ void setsize(u8 const *end) {
    size = end - bytes;
    assert(size <= BUCKETSIZE);
  }
};

template <u32 SIZE>
class twice_set {
  const static u32 TWICE_WORDS = ((2 * SIZE) / 32);
public:
  u32 bits[TWICE_WORDS];
  __device__ void reset() {
    for (u32 b = threadIdx.x; b < TWICE_WORDS; b += blockDim.x)
      bits[b] = 0;
  }
  __device__ void set(node_t u) {
    node_t idx = u/16;
    u32 bit = 1 << (2 * (u%16));
    u32 old = atomicOr(&bits[idx], bit);
    u32 bit2 = bit<<1;
    if ((old & (bit2|bit)) == bit) atomicOr(&bits[idx], bit2);
  }
  __device__ u32 test(node_t u) const {
    return (bits[u/16] >> (2 * (u%16))) & 2;
  }
};

template<u32 BUCKETSIZE, u32 NR, u32 NR1>
struct indexer {
  u32 index[NX];
  zbucket<128,0,0> buf[NY];
  u32 bufindex[NX];
  zbucket<BUCKETSIZE, NR, NR1> (*buckets)[NY];

  __device__ void init(zbucket<BUCKETSIZE, NR, NR1> (*bkts)[NY]) {
    if (!threadIdx.x)
      buckets = bkts;
  }
  __device__ void matrixu(const u32 x) {
    for (u32 y = threadIdx.x; y < NY; y += blockDim.x)
      index[y] = buckets[x][y].bytes - (u8 *)buckets;
  }
  __device__ void matrixv(const u32 y) {
    for (u32 x = threadIdx.x; x < NX; x += blockDim.x)
      index[x] = buckets[x][y].bytes - (u8 *)buckets;
  }
  template <u32 SIZE>
  __device__ void writebig(u32 i, const u64 x) {
    const u32 idx = atomicAdd(index+i, SIZE);
    memcpy((u8 *)buckets + idx, (u8 *)&x, SIZE);
  }
  __device__ void write32(u32 i, const u32 x) {
    const u32 idx = atomicAdd(index+i, sizeof(u32));
    *(u32 *)((u8 *)buckets + idx) = x;
  }
  __device__ void storeu(const u32 x) {
    for (u32 y = threadIdx.x; y < NY; y += blockDim.x)
      buckets[x][y].setsize((u8 *)buckets + index[y]);
  }
  __device__ void storev(const u32 y) {
    for (u32 x = threadIdx.x; x < NX; x += blockDim.x)
      buckets[x][y].setsize((u8 *)buckets + index[x]);
  }
};

#define likely(x)   ((x)!=0)
#define unlikely(x) (x)

class edgetrimmer; // avoid circular references

typedef u8 zbucket8[NYZ1*2];
typedef u32 zbucket32[MAXNZNYZ1];

#define checkCudaErrors(ans) ({ int retval; retval = gpuAssert((ans), __FILE__, __LINE__); retval; })
inline int gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  int device_id;
  hipGetDevice(&device_id);
	//Only spit this to logs once, then flag device to stop trying
  if (code != hipSuccess && !DEVICE_INFO[device_id].threw_error) {
    fprintf(stderr,"Device %d GPUassert: %s %s %d\n", device_id, hipGetErrorString(code), file, line);
    hipDeviceReset();
    mark_device_error(device_id);
    if (abort) return code;
  }
  return 0;
}

typedef u32 proof[PROOFSIZE];

struct twostagetpb {
  u16 stage1tpb;
  u16 stage2tpb;
};

struct trimparams {
  u16 nblocks;
  u16 ntrims;
  u16 genUblocks;
  u16 genUtpb;
  twostagetpb genV;
  twostagetpb trim;
  twostagetpb rename[2];
  u16 trim3tpb;
  u16 rename3tpb;
  u16 reportcount;
  u16 reportrounds;
  
  trimparams() {
    ntrims              = 176;
    nblocks             =  64;
    genUblocks          = 256;
    genUtpb             =   8;
    genV.stage1tpb      =  32;
    genV.stage2tpb      = 128;
    trim.stage1tpb      =  32;
    trim.stage2tpb      = 128;
    rename[0].stage1tpb =  32;
    rename[0].stage2tpb =  64;
    rename[1].stage1tpb =  32;
    rename[1].stage2tpb = 128;
    trim3tpb            =  64;
    rename3tpb          =   2;
    reportcount   = 1;
    reportrounds = 0;
  }
};

// maintains set of trimmable edges
struct edgetrimmer {
  trimparams tp;
  siphash_keys sip_keys;
  edgetrimmer *dt;
  zbucket<ZBUCKETSIZE,NZ1,NZ2> (*buckets)[NY];
  zbucket<TBUCKETSIZE,0,0> (*tbuckets)[NY];
  zbucket32 *tnames;
  u32 *uvnodes;
  proof sol;

  edgetrimmer(const trimparams _tp) {
    tp = _tp; // memcpy(tp, _tp, sizeof(trimparams));
    checkCudaErrors(hipMalloc((void**)&dt, sizeof(edgetrimmer)));
    checkCudaErrors(hipMalloc((void**)&buckets, sizeof(zbucket<ZBUCKETSIZE,NZ1,NZ2>[NX][NY])));
    checkCudaErrors(hipMalloc((void**)&tbuckets, tp.nblocks * sizeof(zbucket<TBUCKETSIZE,0,0>[NY])));
    checkCudaErrors(hipMalloc((void**)&tnames, tp.nblocks * sizeof(zbucket32)));
    checkCudaErrors(hipMalloc((void**)&uvnodes, PROOFSIZE * 2 * sizeof(u32)));
  }
  u64 sharedbytes() const {
    return sizeof(zbucket<ZBUCKETSIZE,NZ1,NZ2>[NX][NY]);
  }
  u64 threadbytes() const {
    return sizeof(zbucket<TBUCKETSIZE,0,0>[NY]) + sizeof(zbucket32);
  }
  ~edgetrimmer() {
    checkCudaErrors(hipFree(buckets));
    checkCudaErrors(hipFree(tbuckets));
    checkCudaErrors(hipFree(tnames));
    checkCudaErrors(hipFree(uvnodes));
  }
  u32 count(u32 size_of) {
    u32 size, sumsize = 0;
    for (u32 ux = 0; ux < tp.reportcount; ux++)
      for (u32 vx = 0; vx < tp.reportcount; vx++) {
        hipMemcpy(&size, &buckets[ux][vx].size, sizeof(u32), hipMemcpyDeviceToHost);
        sumsize += size;
      }
    return sumsize / size_of;
  }

  template <u32 SIZE>
  __device__ void writebig(u8 *p64, const u64 x) {
    memcpy(p64, (u8 *)&x, SIZE);
  }

  __device__ u16 read16(const u8 *p64) {
    u16 foo;
    memcpy((u8 *)&foo, p64, 2);
    return foo;
  }

  template <u32 SIZE>
  __device__ u64 readbig(const u8 *p64) {
    u64 foo = 0;
    memcpy((u8 *)&foo, p64, SIZE);
    return foo;
  }

  __device__ void genUnodes(const u32 uorv) {
    __shared__ indexer<ZBUCKETSIZE,NZ1,NZ2> dst;

    dst.init(buckets);
    for (u32 y = blockIdx.x; y < NY; y += gridDim.x) {
      dst.matrixv(y);
      __syncthreads();
      u32 edge      = y << YZBITS;
      const u32 endedge = edge + NYZ;
      for (edge += threadIdx.x; edge < endedge; edge += blockDim.x) {
// bit        28..21     20..13    12..0
// node       XXXXXX     YYYYYY    ZZZZZ
        const u32 node = dipnode(sip_keys, edge, uorv);
        const u32 ux = node >> YZBITS;
// bit        39..21     20..13    12..0
// write        edge     YYYYYY    ZZZZZ
        dst.writebig<BIGSIZE>(ux, (u64)edge << YZBITS | (node & YZMASK));
      }
      __syncthreads();
      dst.storev(y);
    }
  }

  __device__ void genVnodes1(const u32 part) {
    __shared__ indexer<TBUCKETSIZE,0,0> small;

    small.init(&tbuckets[blockIdx.x]);
    const u32 ux = blockIdx.x + part * gridDim.x;
    {
      small.matrixu(0);
      __syncthreads();
      for (u32 my = 0 ; my < NY; my++) {
        u32 edge = my << YZBITS;
        const u8           *readbg = buckets[ux][my].bytes;
        const u8 * const endreadbg = readbg + buckets[ux][my].size;
// printf("id %d x %d y %d size %u read %d\n", blockIdx.x, ux, my, buckets[ux][my].size, readbg-base);
        for (readbg += BIGSIZE*threadIdx.x; readbg < endreadbg; readbg += BIGSIZE*blockDim.x) {
// bit     39/31..22     21..15    14..0
// read         edge     UYYYYY    UZZZZ   within UX partition
          const u64 e = readbig<BIGSIZE>(readbg);
// u32 oldedge = edge;
	  const u32 lag = NNONYZ >> 2;
          edge += (((u32)(e >> YZBITS) - edge + lag) & (NNONYZ-1)) - lag;
// if (blockIdx.x==4 && edge>oldedge+4096) printf("oldedge %x edge %x delta %d\n",  oldedge, edge, oldedge+NNONYZ-edge);
// if (ux==78 && my==243) printf("id %d ux %d my %d e %08x prefedge %x edge %x\n", blockIdx.x, ux, my, e, e >> YZBITS, edge);
          const u32 uy = (e >> ZBITS) & YMASK;
// bit         39..15     14..0
// write         edge     UZZZZ   within UX UY partition
          small.writebig<BIGSIZE>(uy, ((u64)edge << ZBITS) | (e & ZMASK));;

// printf("id %d ux %d y %d e %010lx e' %010x\n", blockIdx.x, ux, my, e, ((u64)edge << ZBITS) | (e >> YBITS));
        }
        if (unlikely(edge >> NONYZBITS != (((my+1) << YZBITS) - 1) >> NONYZBITS))
        { printf("OOPS1: id %d ux %d y %d edge %x vs %x\n", blockIdx.x, ux, my, edge, ((my+1)<<YZBITS)-1); assert(0); }
      }
      small.storeu(0);
    }
  }

  __device__ void genVnodes2(const u32 part, const u32 uorv) {
    static const u32 NONDEGBITS = (BIGSLOTBITS < 2 * YZBITS ? BIGSLOTBITS : 2 * YZBITS) - ZBITS;
    static const u32 NONDEGMASK = (1 << NONDEGBITS) - 1;
    __shared__ indexer<ZBUCKETSIZE,NZ1,NZ2> dst;
    __shared__ twice_set<NZ> degs;

    dst.init(buckets);
    const u32 ux = blockIdx.x + part * gridDim.x;
    {
      dst.matrixu(ux);
      for (u32 uy = 0 ; uy < NY; uy++) {
        degs.reset();
        __syncthreads();
        u8 *readsmall = tbuckets[blockIdx.x][uy].bytes, *endreadsmall = readsmall + tbuckets[blockIdx.x][uy].size;
// if (blockIdx.x==1) printf("id %d ux %d y %d size %u sumsize %u\n", blockIdx.x, ux, uy, tbuckets[blockIdx.x][uy].size/BIGSIZE, sumsize);
	readsmall += BIGSIZE * threadIdx.x;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall+=BIGSIZE*blockDim.x)
	  degs.set(read16(rdsmall) & ZMASK);
        __syncthreads();
        u32 edge = 0;
	u64 uy34 = (u64)uy << YZZBITS;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall+=BIGSIZE*blockDim.x) {
// bit         39..13     12..0
// read          edge     UZZZZ    sorted by UY within UX partition
          const u64 e = readbig<BIGSIZE>(rdsmall);
// u32 oldedge = edge;
	  const u32 lag = NONDEGMASK >> 2;
          edge += (((e >> ZBITS) - edge + lag) & NONDEGMASK) - lag;
// if (blockIdx.x==4 && edge>oldedge+1000000) printf("oldedge %x edge %x delta %d\n",  oldedge, edge, oldedge+NONDEGMASK+1-edge);
// if (blockIdx.x==0) printf("id %d ux %d uy %d e %010lx pref %4x edge %x mask %x\n", blockIdx.x, ux, uy, e, e>>ZBITS, edge, NONDEGMASK);
	  const u32 z = e & ZMASK;
          if (degs.test(z)) {
            const u32 node = dipnode(sip_keys, edge, uorv);
            const u32 vx = node >> YZBITS; // & XMASK;
// bit        39..34    33..21     20..13     12..0
// write      UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition
            dst.writebig<BIGSIZE>(vx, uy34 | ((u64)z << YZBITS) | (node & YZMASK));
// printf("id %d ux %d y %d edge %08x e' %010lx vx %d\n", blockIdx.x, ux, uy, *readedge, uy34 | ((u64)(node & YZMASK) << ZBITS) | *readz, vx);
	  }
        }
        __syncthreads();
        if (unlikely(edge >> NONDEGBITS != EDGEMASK >> NONDEGBITS))
        { printf("OOPS2: id %d ux %d uy %d edge %x vs %x\n", blockIdx.x, ux, uy, edge, EDGEMASK); assert(0); }
      }
      dst.storeu(ux);
    }
  }

#define mymin(a,b) ((a) < (b) ? (a) : (b))

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimedges1(const u32 round, const u32 part) {
    static const u32 SRCSLOTBITS = mymin(SRCSIZE * 8, 2 * YZBITS);
    static const u32 SRCPREFBITS = SRCSLOTBITS - YZBITS;
    static const u32 SRCPREFMASK = (1 << SRCPREFBITS) - 1;
    __shared__ indexer<TBUCKETSIZE,0,0> small;

    small.init(&tbuckets[blockIdx.x]);
    const u32 vx = blockIdx.x + part * gridDim.x;
    {
      small.matrixu(0);
      for (u32 ux = 0; ux < NX; ux++) {
        __syncthreads();
        u32 uyz = 0;
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        const u8 *readbg = zb.bytes;
        const u8 * const endreadbg = readbg + zb.size;
// if (!blockIdx.x && !threadIdx.x)
// printf("round %d vx %d ux %d size %u\n", round, vx, ux, pzb->size/SRCSIZE);
        for (readbg += SRCSIZE*threadIdx.x; readbg < endreadbg; readbg += SRCSIZE*blockDim.x) {
// bit     43/39..37    36..22     21..15     14..0
// write      UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition
          const u64 e = readbig<SRCSIZE>(readbg); // & SRCSLOTMASK;
// if (!blockIdx.x && !threadIdx.x && round==4 && ux+vx==0)
// printf("id %d vx %d ux %d e %010llx suffUXYZ %05x suffUXY %03x UXYZ %08x UXY %04x mask %x\n", blockIdx.x, vx, ux, e, (u32)(e >> YZBITS), (u32)(e >> YZZBITS), uxyz, uxyz>>ZBITS, SRCPREFMASK);

	  const u32 lag = SRCPREFMASK >> 2;
          if (SRCPREFBITS >= YZBITS)
	    uyz = e >> YZBITS;
	  else uyz += (((u32)(e >> YZBITS) - uyz + lag) & SRCPREFMASK) - lag;
          const u32 vy = (e >> ZBITS) & YMASK;
// if (round==12)
//     printf("id %d.%d vx %d vy %d e1 %010lx e %010lx suffUX %02x UX %x\n", blockIdx.x, threadIdx.x, vx, vy, e1 , e, (u32)(e >> YZZBITS), ux);
// bit     43/39..37    36..30     29..15     14..0
// write      UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition
          small.writebig<DSTSIZE>(vy, ((u64)(ux << YZBITS | uyz) << ZBITS) | (e & ZMASK));
          uyz &= ~ZMASK;
        }
        if (unlikely(uyz >> ZBITS >= NY))
        { printf("OOPS3: id %d vx %d ux %d uyz %x\n", blockIdx.x, vx, ux, uyz); break; }
      }
      small.storeu(0);
    }
  }

  template <u32 DSTSIZE, bool TRIMONV>
  __device__ void trimedges2(const u32 round, const u32 part) {
    static const u32 DSTSLOTBITS = mymin(DSTSIZE * 8, 2 * YZBITS);
    static const u32 DSTPREFBITS = DSTSLOTBITS - YZZBITS;
    static const u32 DSTPREFMASK = (1 << DSTPREFBITS) - 1;
    __shared__ indexer<ZBUCKETSIZE,NZ1,NZ2> dst;
    __shared__ twice_set<NZ> degs;

    dst.init(buckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    {
      TRIMONV ? dst.matrixv(vx) : dst.matrixu(vx);
      for (u32 vy = 0 ; vy < NY; vy++) {
        const u64 vy34 = (u64)vy << YZZBITS;
        degs.reset();
        __syncthreads();
        u8 *readsmall = tbuckets[blockIdx.x][vy].bytes, *endreadsmall = readsmall + tbuckets[blockIdx.x][vy].size;
// printf("id %d vx %d vy %d size %u sumsize %u\n", blockIdx.x, vx, vy, tbuckets[blockIdx.x][vx].size/BIGSIZE, sumsize);
        readsmall += DSTSIZE * threadIdx.x;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall += DSTSIZE*blockDim.x)
	  degs.set(read16(rdsmall) & ZMASK);
        __syncthreads();
        u32 ux = 0;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall += DSTSIZE*blockDim.x) {
// bit     41/39..34    33..26     25..13     12..0
// read       UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition
// bit     45/39..37    36..30     29..15     14..0      with XBITS==YBITS==7
// read       UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition
          const u64 e = readbig<DSTSIZE>(rdsmall); //  & DSTSLOTMASK;
	  const u32 lag = DSTPREFMASK >> 2;
          ux += (((u32)(e >> YZZBITS) - ux + lag) & DSTPREFMASK) - lag;
// if (round==12 && vx==0x49 && (e==0xec46dd5fa5ULL || e==0xed023593c3ULL || e==0xee6743a841ULL
//    || e==0xece4d1f4b3ULL || e==0xed26caec88ULL || e==0xf8523e9becULL))
//  printf("id %d.%d vx %d vy %d e %010lx suffUX %02x UX %x mask %x\n", blockIdx.x, threadIdx.x, vx, vy, e, (u32)(e >> YZZBITS), ux, DSTPREFMASK);
// bit    41/39..34    33..21     20..13     12..0
// write     VYYYYY    VZZZZZ     UYYYYY     UZZZZ   within UX partition
          if (degs.test(e & ZMASK))
            dst.writebig<DSTSIZE>(ux, vy34 | ((e & ZMASK) << YZBITS) | ((e >> ZBITS) & YZMASK));
        }
        __syncthreads();
        if (unlikely(ux >> DSTPREFBITS != XMASK >> DSTPREFBITS))
        { printf("OOPS4: id %d.%d vx %x ux %x vs %x\n", blockIdx.x, threadIdx.x, vx, ux, XMASK); }
      }
      TRIMONV ? dst.storev(vx) : dst.storeu(vx);
    }
  }

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimrename1(const u32 round, const u32 part) {
    static const u32 SRCSLOTBITS = mymin(SRCSIZE * 8, (TRIMONV ? YZBITS : YZ1BITS) + YZBITS);
    static const u32 SRCPREFBITS = SRCSLOTBITS - YZBITS;
    static const u32 SRCPREFMASK = (1 << SRCPREFBITS) - 1;
    __shared__ indexer<TBUCKETSIZE,0,0> small;

    small.init(&tbuckets[blockIdx.x]);
    const u32 vx = blockIdx.x + part * gridDim.x;
    {
      small.matrixu(0);
      for (u32 ux = 0 ; ux < NX; ux++) {
        __syncthreads();
        u32 uyz = 0;
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        const u8 *readbg = zb.bytes;
	const u8 * const endreadbg = readbg + zb.size;
// printf("id %d vx %d ux %d size %u\n", blockIdx.x, vx, ux, zb.size/SRCSIZE);
        for (readbg += SRCSIZE*threadIdx.x; readbg < endreadbg; readbg += SRCSIZE*blockDim.x) {
// bit        39..37    36..22     21..15     14..0
// write      UYYYYY    UZZZZZ     VYYYYY     VZZZZ   within VX partition  if TRIMONV
// bit            36...22     21..15     14..0
// write          VYYYZZ'     UYYYYY     UZZZZ   within UX partition  if !TRIMONV
          const u64 e = readbig<SRCSIZE>(readbg); //  & SRCSLOTMASK;
	  const u32 lag = SRCPREFMASK >> 2;
          if (TRIMONV)
            uyz += (((u32)(e >> YZBITS) - uyz + lag) & SRCPREFMASK) - lag;
          else uyz = e >> YZBITS;
// if (round==32 && ux==25) printf("id %d vx %d ux %d e %010lx suffUXYZ %05x suffUXY %03x UXYZ %08x UXY %04x mask %x\n", blockIdx.x, vx, ux, e, (u32)(e >> YZBITS), (u32)(e >> YZZBITS), uxyz, uxyz>>ZBITS, SRCPREFMASK);
          const u32 vy = (e >> ZBITS) & YMASK;
// bit        39..37    36..30     29..15     14..0
// write      UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition  if TRIMONV
// bit            36...30     29...15     14..0
// write          VXXXXXX     VYYYZZ'     UZZZZ   within UX UY partition  if !TRIMONV
          small.writebig<SRCSIZE>(vy, ((u64)(ux << (TRIMONV ? YZBITS : YZ1BITS) | uyz) << ZBITS) | (e & ZMASK));
// if (TRIMONV&&vx==75&&vy==83) printf("id %d vx %d vy %d e %010lx e15 %x ux %x\n", blockIdx.x, vx, vy, ((u64)uxyz << ZBITS) | (e & ZMASK), uxyz, uxyz>>YZBITS);
          if (TRIMONV)
            uyz &= ~ZMASK;
        }
      }
      small.storeu(0);
    }
  }

  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
  __device__ void trimrename2(const u32 round, const u32 part) {
    static const u32 SRCSLOTBITS = mymin(SRCSIZE * 8, (TRIMONV ? YZBITS : YZ1BITS) + YZBITS);
    static const u32 SRCPREFBITS2 = SRCSLOTBITS - YZZBITS;
    static const u32 SRCPREFMASK2 = (1 << SRCPREFBITS2) - 1;
    __shared__ indexer<ZBUCKETSIZE,NZ1,NZ2> dst;
    __shared__ twice_set<NZ> degs;
    const u32 NONAME = ~0;
    u32 maxrename = 0;

    dst.init(buckets);
    const u32 vx = blockIdx.x + part * gridDim.x;
    {
      TRIMONV ? dst.matrixv(vx) : dst.matrixu(vx);
      u32 *names = tnames[blockIdx.x];
      u32 nrenames = threadIdx.x;
      for (u32 vy = 0 ; vy < NY; vy++) {
        for (u32 z = threadIdx.x; z < NZ; z += blockDim.x)
          names[z] = NONAME;
        degs.reset();
        __syncthreads();
        u8    *readsmall = tbuckets[blockIdx.x][vy].bytes, *endreadsmall = readsmall + tbuckets[blockIdx.x][vy].size;
// printf("id %d vx %d vy %d size %u sumsize %u\n", blockIdx.x, vx, vy, tbuckets[blockIdx.x][vx].size/BIGSIZE, sumsize);
        readsmall += SRCSIZE * threadIdx.x;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall += SRCSIZE*blockDim.x)
	  degs.set(read16(rdsmall) & ZMASK);
        __syncthreads();
        u32 ux = 0;
        for (u8 *rdsmall = readsmall; rdsmall < endreadsmall; rdsmall += SRCSIZE*blockDim.x) {
// bit     41/39..34    33..26     25..13     12..0
// read       UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition  if TRIMONV and XBITS==8
// bit        39..37    36..30     29..15     14..0
// read       UXXXXX    UYYYYY     UZZZZZ     VZZZZ   within VX VY partition  if TRIMONV
// bit            36...30     29...15     14..0
// read           VXXXXXX     VYYYZZ'     UZZZZ   within UX UY partition  if !TRIMONV
          const u64 e = readbig<SRCSIZE>(rdsmall); //  & SRCSLOTMASK;
	  const u32 lag = SRCPREFMASK2 >> 2;
          if (TRIMONV) {
            if (SRCPREFBITS2 >= XBITS)
	      ux = e >> YZZBITS;
	    else ux += (((u32)(e >> YZZBITS) - ux + lag) & SRCPREFMASK2) - lag;
	  } else ux = e >> YZZ1BITS;
          const u32 vz = e & ZMASK;
// if (TRIMONV&&vx==135&&vy==147) printf("id %d vx %d vy %d e %012llx e37 %x ux %x vz %d nrenames %d\n", threadIdx.x, vx, vy, e, (u32)(e>>YZZBITS), ux, vz, nrenames);
          if (degs.test(vz)) {
            u32 vdeg = atomicCAS(&names[vz], NONAME, nrenames);
            if (vdeg == NONAME) {
              vdeg = nrenames;
              if (TRIMONV)
	        buckets[vdeg >> Z1BITS][vx].renamev[vdeg & Z1MASK] = vy << ZBITS | vz;
	      else
	        buckets[vx][vdeg >> Z1BITS].renameu[vdeg & Z1MASK] = vy << ZBITS | vz;
	      nrenames += blockDim.x;
            }
// bit       36..22     21..15     14..0
// write     VYYZZ'     UYYYYY     UZZZZ   within UX partition  if TRIMONV
            if (TRIMONV)
               dst.writebig<DSTSIZE>(ux, ((u64)vdeg << YZBITS ) | ((e >> ZBITS) & YZMASK));
            else dst.write32(ux, (vdeg << YZ1BITS) | ((e >> ZBITS) & YZ1MASK));
// if (vx==44&&vy==58) printf("  id %d vx %d vy %d newe %010lx\n", blockIdx.x, vx, vy, vy28 | ((vdeg) << YZBITS) | ((e >> ZBITS) & YZMASK));
          }
        }
        __syncthreads();
        if (TRIMONV && unlikely(ux >> SRCPREFBITS2 != XMASK >> SRCPREFBITS2))
        { printf("OOPS6: id %d vx %d vy %d ux %x vs %x\n", blockIdx.x, vx, vy, ux, XMASK); break; }
      }
      if (nrenames > maxrename)
        maxrename = nrenames;
      TRIMONV ? dst.storev(vx) : dst.storeu(vx);
    }
    // if (!blockIdx.x && !threadIdx.x ) printf("maxrename %d\n", maxrename);
    assert(maxrename < NYZ1);
  }

  template <bool TRIMONV>
  __device__ void trimedges3(const u32 round) {
    __shared__ twice_set<NYZ1> degs;

    // if (!blockIdx.x && !threadIdx.x ) printf("%dx%d threads\n", gridDim.x, blockDim.x);
    for (u32 vx = blockIdx.x; vx < NY; vx += gridDim.x) {
      __syncthreads();
      degs.reset();
      __syncthreads();
      for (u32 ux = threadIdx.x ; ux < NX; ux += blockDim.x) {
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        // printf("id %d vx %d ux %d size %d\n", blockIdx.x, vx, ux, zb.size/SRCSIZE);
        for (; readbg < endreadbg; readbg++)
          degs.set(*readbg & YZ1MASK);
      }
      __syncthreads();
      for (u32 ux = threadIdx.x ; ux < NX; ux += blockDim.x) {
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        u32 *bwrite = readbg;
        for (; readbg < endreadbg; readbg++) {
// bit       29..23    22..15     14..8     7..0
// read      UYYYYY    UZZZZ'     VYYYY     VZZ'   within VX partition
          const u32 e = *readbg;
          const u32 vyz = e & YZ1MASK;
// bit       29..23    22..15     14..8     7..0
// write     VYYYYY    VZZZZ'     UYYYY     UZZ'   within UX partition
          if (degs.test(vyz))
            *bwrite++ = (vyz << YZ1BITS) | (e >> YZ1BITS);
// if (e==0x30951) printf("round %d id %d\ntest %d vx %d ux %d e %08x readbig %d\n", round, threadIdx.x, degs.test(vyz), vx, ux, e, readbg-zb.words);
        }
        zb.setsize((u8 *)bwrite);
      }
    }
  }

  template <bool TRIMONV>
  __device__ void trimrename3(const u32 round) {
    __shared__ twice_set<NYZ1> degs;
    const u32 NONAME = ~0;
    u32 maxrename = 0;

    u32 *names = tnames[blockIdx.x];
    for (u32 vx = blockIdx.x; vx < NY; vx += gridDim.x) {
      __syncthreads();
      for (u32 z = threadIdx.x; z < NYZ1; z += blockDim.x)
        names[z] = NONAME;
      degs.reset();
      __syncthreads();
      for (u32 ux = threadIdx.x; ux < NX; ux += blockDim.x) {
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        // printf("id %d vx %d ux %d size %d\n", blockIdx.x, vx, ux, zb.size/SRCSIZE);
        for (; readbg < endreadbg; readbg ++)
          degs.set(*readbg & YZ1MASK);
      }
      u32 nrenames = threadIdx.x;
      __syncthreads();
      for (u32 ux = threadIdx.x; ux < NX; ux += blockDim.x) {
        zbucket<ZBUCKETSIZE,NZ1,NZ2> &zb = TRIMONV ? buckets[ux][vx] : buckets[vx][ux];
        zbucket<Z2BUCKETSIZE,0,0> &wb = ((zbucket<Z2BUCKETSIZE,0,0> (*)[NY])tbuckets)[vx][ux];
        u32 *readbg = zb.words, *endreadbg = readbg + zb.size/sizeof(u32);
        u32 *bwrite = TRIMONV ? readbg : wb.words;
        for (; readbg < endreadbg; readbg ++) {
// bit       29...15     14...0
// read      UYYYZZ'     VYYZZ'   within VX partition
          const u32 e = *readbg;
          const u32 vyz = e & YZ1MASK;
          if (degs.test(vyz)) {
            u32 vdeg = atomicCAS(&names[vyz], NONAME, nrenames);
            if (vdeg == NONAME) {
              vdeg = nrenames;
              if (TRIMONV)
	        buckets[vdeg >> Z2BITS][vx].renamev1[vdeg & Z2MASK] = vyz;
	      else
	        buckets[vx][vdeg >> Z2BITS].renameu1[vdeg & Z2MASK] = vyz;
	      nrenames += blockDim.x;
#ifdef CHECKNAMES
	      if (nrenames >= NYZ2) {
                printf("OUT OF NAMES! Dropping %d edges from ux %d vx %d\n", (u32)(endreadbg-readbg-1), ux, vx);
                break;
              }
#endif
            }
// bit       23...15     14...0
// write     VYYZZZ"     UYYZZ'   within UX partition
            if (!TRIMONV && bwrite-wb.words >= Z2BUCKETSIZE/sizeof(u32)) {
              printf("BUCKET FULL! Dropping %d edges from ux %d vx %d\n", (u32)(endreadbg-readbg), ux, vx);
              break;
            }
            *bwrite++ = (vdeg << (TRIMONV ? YZ1BITS : YZ2BITS)) | (e >> YZ1BITS);
          }
        }
        if (TRIMONV) zb.size = (u8 *)bwrite - zb.bytes;
	else wb.setsize((u8 *)bwrite);
      }
      if (nrenames > maxrename)
        maxrename = nrenames;
    }
    assert(maxrename < NYZ2);
  }

  __device__ void recoveredges() {
    __shared__ u32 u, ux, uyz, v, vx, vyz;

    if (threadIdx.x == 0) {
      const u32 u1 = uvnodes[2*blockIdx.x], v1 = uvnodes[2*blockIdx.x+1];
      ux = u1 >> YZ2BITS;
      vx = v1 >> YZ2BITS;
      uyz = buckets[ux][(u1 >> Z2BITS) & YMASK].renameu1[u1 & Z2MASK];
      assert(uyz < NYZ1);
      vyz = buckets[(v1 >> Z2BITS) & YMASK][vx].renamev1[v1 & Z2MASK];
      assert(vyz < NYZ1);
#if COMPRESSROUND > 0
      uyz = buckets[ux][uyz >> Z1BITS].renameu[uyz & Z1MASK];
      vyz = buckets[vyz >> Z1BITS][vx].renamev[vyz & Z1MASK];
#endif
      u = (ux << YZBITS) | uyz;
      v = (vx << YZBITS) | vyz;
      uvnodes[2*blockIdx.x] = u;
      uvnodes[2*blockIdx.x+1] = v;
    }
    __syncthreads();
  }

  __device__ void recoveredges1() {
    __shared__ u32 uxymap[NXY/32];

    for (u32 i = threadIdx.x; i < PROOFSIZE; i += blockDim.x) {
      const u32 uxy = uvnodes[2*i] >> ZBITS;
      atomicOr(&uxymap[uxy/32], 1 << uxy%32);
    }
    __syncthreads();
    for (u32 edge = blockIdx.x * blockDim.x + threadIdx.x; edge < NEDGES; edge += gridDim.x * blockDim.x) {
      const u32 u = dipnode(sip_keys, edge, 0);
      const u32 uxy = u  >> ZBITS;
      if ((uxymap[uxy/32] >> uxy%32) & 1) {
	for (u32 j = 0; j < PROOFSIZE; j++) {
           if (uvnodes[2*j] == u && dipnode(sip_keys, edge, 1) == uvnodes[2*j+1]) {
             sol[j] = edge;
           }
        }
      }
    }
  }

  int trim();
  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV> void _trimedges(edgetrimmer *et, const u32 round);
  template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV> void _trimrename(edgetrimmer *et, const u32 round);
};

__global__ void _genUnodes(edgetrimmer *et, const u32 uorv) {
  et->genUnodes(uorv);
}

__global__ void _genVnodes1(edgetrimmer *et, const u32 part) {
  et->genVnodes1(part);
}

__global__ void _genVnodes2(edgetrimmer *et, const u32 part, const u32 uorv) {
  et->genVnodes2(part, uorv);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimedges1(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimedges1<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 DSTSIZE, bool TRIMONV>
__global__ void _trimedges2(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimedges2<DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
void edgetrimmer::_trimedges(edgetrimmer *et, const u32 round) {
  for (u32 part=0; part < NX/tp.nblocks; part++) {
    _trimedges1<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.trim.stage1tpb>>>(dt, round, part);
    _trimedges2<         DSTSIZE, TRIMONV><<<tp.nblocks,tp.trim.stage2tpb>>>(dt, round, part);
  }
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimrename1(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimrename1<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
__global__ void _trimrename2(edgetrimmer *et, const u32 round, const u32 part) {
  et->trimrename2<SRCSIZE, DSTSIZE, TRIMONV>(round, part);
}

template <u32 SRCSIZE, u32 DSTSIZE, bool TRIMONV>
void edgetrimmer::_trimrename(edgetrimmer *et, const u32 round) {
  for (u32 part=0; part < NX/tp.nblocks; part++) {
    _trimrename1<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.rename[round&1].stage1tpb>>>(dt, round, part);
    _trimrename2<SRCSIZE, DSTSIZE, TRIMONV><<<tp.nblocks,tp.rename[round&1].stage2tpb>>>(dt, round, part);
  }
}

template <bool TRIMONV>
__global__ void _trimedges3(edgetrimmer *et, const u32 round) {
  et->trimedges3<TRIMONV>(round);
}

template <bool TRIMONV>
__global__ void _trimrename3(edgetrimmer *et, const u32 round) {
  et->trimrename3<TRIMONV>(round);
}

__global__ void _recoveredges(edgetrimmer *et) {
  et->recoveredges();
}

__global__ void _recoveredges1(edgetrimmer *et) {
  et->recoveredges1();
}

#ifndef EXPANDROUND
#define EXPANDROUND 5
#endif

#if EXPANDROUND < COMPRESSROUND
#define BIGGERSIZE BIGSIZE+1
#else
#define BIGGERSIZE BIGSIZE
#endif

  int edgetrimmer::trim() {
    hipMemcpy(dt, this, sizeof(edgetrimmer), hipMemcpyHostToDevice);
    hipEvent_t start, stop, startall, stopall;
    if (checkCudaErrors(hipEventCreate(&startall))) return 0; if (checkCudaErrors(hipEventCreate(&stopall))) return 0;
    hipEventRecord(startall, NULL);
    if (checkCudaErrors(hipEventCreate(&start))) return 0; if (checkCudaErrors(hipEventCreate(&stop))) return 0;
    float duration;
    hipEventRecord(start, NULL);
    _genUnodes<<<tp.genUblocks,tp.genUtpb>>>(dt, 0);
    if (checkCudaErrors(hipDeviceSynchronize())) return 0; hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (0 < tp.reportrounds)
      printf("genUnodes size %u completed in %.0f ms\n", count(BIGSIZE), duration);
    hipEventRecord(start, NULL);
    for (u32 part=0; part < NX/tp.nblocks; part++) {
      _genVnodes1<<<tp.nblocks,tp.genV.stage1tpb>>>(dt, part);
      _genVnodes2<<<tp.nblocks,tp.genV.stage2tpb>>>(dt, part, 1);
    }
    checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
    hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    if (1 < tp.reportrounds)
      printf("genVnodes size %u completed in %.0f ms\n", count(BIGSIZE), duration);
    for (u32 round = 2; round < tp.ntrims-2; round += 2) {
      hipEventRecord(start, NULL);
      u32 size_of = BIGGERSIZE;
      if (round < COMPRESSROUND) {
        if (round < EXPANDROUND) {
          _trimedges<BIGSIZE, BIGSIZE, true>(dt, round);
          size_of = BIGSIZE;
	} else if (round == EXPANDROUND) {
          _trimedges<BIGSIZE, BIGGERSIZE, true>(dt, round);
	} else _trimedges<BIGGERSIZE, BIGGERSIZE, true>(dt, round);
      } else if (round==COMPRESSROUND) {
        _trimrename<BIGGERSIZE, BIGGERSIZE, true>(dt, round);
      } else {
        _trimedges3<true><<<tp.nblocks,tp.trim3tpb>>>(dt, round);
        size_of = sizeof(u32);
      }
      checkCudaErrors(hipDeviceSynchronize()); hipEventRecord(stop, NULL);
      hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
      if (round < tp.reportrounds)
        printf("round %d size %u completed in %.0f ms\n", round, count(size_of), duration);

      hipEventRecord(start, NULL);
      size_of = BIGGERSIZE;
      if (round < COMPRESSROUND) {
        if (round+1 < EXPANDROUND) {
          _trimedges<BIGSIZE, BIGSIZE, false>(dt, round+1);
          size_of = BIGGERSIZE;
	} else if (round+1 == EXPANDROUND) {
          _trimedges<BIGSIZE, BIGGERSIZE, false>(dt, round+1);
	} else _trimedges<BIGGERSIZE, BIGGERSIZE, false>(dt, round+1);
      } else if (round==COMPRESSROUND) {
        _trimrename<BIGGERSIZE, sizeof(u32), false>(dt, round+1);
        size_of = sizeof(u32);
      } else {
        _trimedges3<false><<<tp.nblocks,tp.trim3tpb>>>(dt, round+1);
        size_of = sizeof(u32);
      }
      if (checkCudaErrors(hipDeviceSynchronize())) return 0; hipEventRecord(stop, NULL);
      hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
      if (round+1 < tp.reportrounds)
        printf("round %d size %u completed in %.0f ms\n", round+1, count(size_of), duration);
    }

    // hipEventRecord(start, NULL);
    _trimrename3<true ><<<tp.nblocks,tp.rename3tpb>>>(dt, tp.ntrims-2);
    if (checkCudaErrors(hipDeviceSynchronize())) return 0; // hipEventRecord(stop, NULL);
    // hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    // printf("rename3 size %u completed in %.0f ms\n", count(cnt,cnt), duration);

    // hipEventRecord(start, NULL);
    _trimrename3<false><<<tp.nblocks,tp.rename3tpb>>>(dt, tp.ntrims-1);
    if (checkCudaErrors(hipDeviceSynchronize())) return 0; // hipEventRecord(stop, NULL);
    // hipEventSynchronize(stop); hipEventElapsedTime(&duration, start, stop);
    // printf("rename3 size %u completed in %.0f ms\n", count(cnt,cnt), duration);

    hipEventRecord(stopall, NULL); hipEventSynchronize(stopall); hipEventElapsedTime(&duration, startall, stopall);
    printf("trim completed in %.0f ms\n", duration);
  }

#define NODEBITS (EDGEBITS + 1)

// grow with cube root of size, hardly affected by trimming
const static u32 MAXPATHLEN = 8 << ((NODEBITS+2)/3);

const static u32 CUCKOO_SIZE = 2 * NX * NYZ2;

int nonce_cmp(const void *a, const void *b) {
  return *(u32 *)a - *(u32 *)b;
}

class solver_ctx {
public:
  edgetrimmer *trimmer;
  zbucket<Z2BUCKETSIZE,0,0> (*buckets)[NY];
  u32 *cuckoo;
  u32 uvnodes[2*PROOFSIZE];
  std::bitset<NXY> uxymap;
  std::vector<u32> sols; // concatanation of all proof's indices

  solver_ctx(const trimparams tp) {
    trimmer = new edgetrimmer(tp);
    cuckoo = 0;
  }
  void setheadernonce(char* const headernonce, const u32 len, const u32 nonce) {
    ((u32 *)headernonce)[len/sizeof(u32)-1] = htole32(nonce); // place nonce at end
    setheader(headernonce, len, &trimmer->sip_keys);
    sols.clear();
  }
  void setheadergrin(const char* header, const u32 len) {
    setheader(header, len, &trimmer->sip_keys);
    sols.clear();
  }
  ~solver_ctx() {
    delete trimmer;
  }

  void recordedge(const u32 i, const u32 u2, const u32 v2) {
    uvnodes[2*i]   = u2/2;
    uvnodes[2*i+1] = v2/2;
  }

  void solution(const u32 *us, u32 nu, const u32 *vs, u32 nv) {
    u32 ni = 0;
    recordedge(ni++, *us, *vs);
    while (nu--)
      recordedge(ni++, us[(nu+1)&~1], us[nu|1]); // u's in even position; v's in odd
    while (nv--)
    recordedge(ni++, vs[nv|1], vs[(nv+1)&~1]); // u's in odd position; v's in even
    assert(ni == PROOFSIZE);
    sols.resize(sols.size() + PROOFSIZE);
    hipMemcpy(trimmer->uvnodes, uvnodes, sizeof(uvnodes), hipMemcpyHostToDevice);
    _recoveredges<<<PROOFSIZE,1>>>(trimmer->dt);
    _recoveredges1<<<4096,128>>>(trimmer->dt);
    hipMemcpy(&sols[sols.size() - PROOFSIZE], trimmer->dt->sol, sizeof(trimmer->sol), hipMemcpyDeviceToHost);
    qsort(&sols[sols.size()-PROOFSIZE], PROOFSIZE, sizeof(u32), nonce_cmp);
  }

  static const u32 CUCKOO_NIL = ~0;

  u32 path(u32 u, u32 *us) const {
    u32 nu, u0 = u;
    for (nu = 0; u != CUCKOO_NIL; u = cuckoo[u]) {
      if (nu >= MAXPATHLEN) {
        while (nu-- && us[nu] != u) ;
        if (!~nu)
          printf("maximum path length exceeded\n");
        else printf("illegal %4d-cycle from node %d\n", MAXPATHLEN-nu, u0);
        exit(0);
      }
      us[nu++] = u;
    }
    return nu-1;
  }

  void findcycles() {
    u32 us[MAXPATHLEN], vs[MAXPATHLEN];

    u32 sumsize = 0;
    for (u32 vx = 0; vx < NX; vx++) {
      for (u32 ux = 0 ; ux < NX; ux++) {
        zbucket<Z2BUCKETSIZE,0,0> &zb = buckets[ux][vx];
	u32 size = zb.size / sizeof(u32);
        u32 *readbg = zb.words, *endreadbg = readbg + size;
	sumsize += size;
        for (; readbg < endreadbg; readbg++) {
// bit        21..11     10...0
// write      UYYZZZ'    VYYZZ'   within VX partition
          const u32 e = *readbg;
	  assert(e < NYZ2*NYZ2);
          const u32 uxyz = (ux << YZ2BITS) | (e >> YZ2BITS);
          const u32 vxyz = (vx << YZ2BITS) | (e & YZ2MASK);
          const u32 u0 = uxyz << 1, v0 = (vxyz << 1) | 1;
          if (u0 != CUCKOO_NIL) {
            u32 nu = path(u0, us), nv = path(v0, vs);
// printf("vx %02x ux %02x e %08x uxyz %06x vxyz %06x u0 %x v0 %x nu %d nv %d\n", vx, ux, e, uxyz, vxyz, u0, v0, nu, nv);
            if (us[nu] == vs[nv]) {
              const u32 min = nu < nv ? nu : nv;
              for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
              const u32 len = nu + nv + 1;
              printf("%4d-cycle found\n", len);
              if (len == PROOFSIZE)
                solution(us, nu, vs, nv);
            } else if (nu < nv) {
              while (nu--)
                cuckoo[us[nu+1]] = us[nu];
              cuckoo[u0] = v0;
            } else {
              while (nv--)
                cuckoo[vs[nv+1]] = vs[nv];
              cuckoo[v0] = u0;
            }
          }
        }
      }
    }
    printf("findcycles completed on %d edges\n", sumsize);
  }

  int solve() {
    if (!trimmer->trim()) return 0;
    buckets = new zbucket<Z2BUCKETSIZE,0,0>[NX][NY];
    printf("start hipMemcpy\n");
    if (checkCudaErrors(hipMemcpy(buckets, trimmer->tbuckets, sizeof(zbucket<Z2BUCKETSIZE,0,0>[NX][NY]), hipMemcpyDeviceToHost))) {
        delete[] buckets;
        return 0;
    }
    printf("end hipMemcpy\n");
    cuckoo = new u32[CUCKOO_SIZE];
    memset(cuckoo, (int)CUCKOO_NIL, CUCKOO_SIZE * sizeof(u32));
    findcycles();
    delete[] cuckoo;
    delete[] buckets;
    return sols.size() / PROOFSIZE;
  }
};

#include <unistd.h>

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

extern "C" int cuckoo_call(char* header_data,
                           int header_length,
                           u32* sol_nonces ) {
  u64 start_time=timestamp();
  trimparams tp;
  u32 nonce = 0;
  u32 range = 1;
  u32 device = 0;
  char header[HEADERLEN];
  u32 len, timems;
  struct timeval time0, time1;
  /*int c;
	  memset(header, 0, sizeof(header));
    while ((c = getopt(argc, argv, "sb:c:d:h:k:m:n:r:U:u:V:v:T:t:X:x:Y:y:Z:z:")) != -1) {
    switch (c) {
      case 's':
        printf("SYNOPSIS\n  cuda30 [-b sblocks] [-c count] [-d device] [-h hexheader] [-k rounds] [-m trims] [-n nonce] [-r range] [-U sblocks] [-u threads] [-V threads] [-v threads] [-T threads] [-t threads] [-X threads] [-x threads] [-Y threads] [-y threads] [-Z threads] [-z threads]\n");
        printf("DEFAULTS\n  cuda30 -b %d -c %d -d %d -h \"\" -k %d -m %d -n %d -r %d -U %d -u %d -V %d -v %d -T %d -t %d -X %d -x %d -Y %d -y 128 -Z 64 -z 2\n", tp.nblocks, tp.reportcount, device, tp.reportrounds, tp.ntrims, nonce, range, tp.genUblocks, tp.genUtpb, tp.genV.stage1tpb, tp.genV.stage2tpb, tp.trim.stage1tpb, tp.trim.stage2tpb, tp.rename[0].stage1tpb, tp.rename[0].stage2tpb, tp.rename[1].stage1tpb, tp.rename[1].stage1tpb, tp.trim3tpb, tp.rename3tpb);
        exit(0);
      case 'b':
        tp.nblocks = atoi(optarg);
        break;
      case 'd':
        device = atoi(optarg);
        break;
      case 'k':
        tp.reportrounds = atoi(optarg);
        break;
      case 'l':
        tp.reportcount = atoi(optarg);
        break;
      case 'h':
        len = strlen(optarg)/2;
        assert(len <= sizeof(header));
        for (u32 i=0; i<len; i++)
          sscanf(optarg+2*i, "%2hhx", header+i);
        break;
      case 'n':
        nonce = atoi(optarg);
        break;
      case 'm':
        tp.ntrims = atoi(optarg) & -2; // make even as required by solve()
        break;
      case 'r':
        range = atoi(optarg);
        break;
      case 'U':
        tp.genUblocks = atoi(optarg);
        break;
      case 'u':
        tp.genUtpb = atoi(optarg);
        break;
      case 'V':
        tp.genV.stage1tpb = atoi(optarg);
        break;
      case 'v':
        tp.genV.stage2tpb = atoi(optarg);
        break;
      case 'T':
        tp.trim.stage1tpb = atoi(optarg);
        break;
      case 't':
        tp.trim.stage2tpb = atoi(optarg);
        break;
      case 'X':
        tp.rename[0].stage1tpb = atoi(optarg);
        break;
      case 'x':
        tp.rename[0].stage2tpb = atoi(optarg);
        break;
      case 'Y':
        tp.rename[1].stage1tpb = atoi(optarg);
        break;
      case 'y':
        tp.rename[1].stage2tpb = atoi(optarg);
        break;
      case 'Z':
        tp.trim3tpb = atoi(optarg);
        break;
      case 'z':
        tp.rename3tpb = atoi(optarg);
        break;
    }
  }*/

  /*int nDevices;
  hipGetDeviceCount(&nDevices);
  assert(device < nDevices);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);
  u64 dbytes = prop.totalGlobalMem;
  int dunit;
  for (dunit=0; dbytes >= 10240; dbytes>>=10,dunit++) ;
  printf("%s with %d%cB @ %d bits x %dMHz\n", prop.name, (u32)dbytes, " KMGT"[dunit], prop.memoryBusWidth, prop.memoryClockRate/1000);
  hipSetDevice(device);*/

  printf("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
  if (range > 1)
    printf("-%d", nonce+range-1);
  printf(") with 50%% edges, %d*%d buckets, %d trims, and %d thread blocks.\n", NX, NY, tp.ntrims, tp.nblocks); 

  solver_ctx ctx(tp);

  u64 sbytes = ctx.trimmer->sharedbytes();
  u64 tbytes = ctx.trimmer->threadbytes();
  u64 bytes = sbytes + tp.nblocks * tbytes;
  int sunit,tunit,unit;
  for (sunit=0; sbytes >= 10240; sbytes>>=10,sunit++) ;
  for (tunit=0; tbytes >= 10240; tbytes>>=10,tunit++) ;
  for ( unit=0;  bytes >= 10240;  bytes>>=10, unit++) ;
  printf("Using %d%cB bucket memory and %d%cB memory per thread block (%d%cB total)\n",
    sbytes, " KMGT"[sunit], tbytes, " KMGT"[tunit], bytes, " KMGT"[unit], NX);

  u32 sumnsols = 0;
  for (int r = 0; r < range; r++) {
    gettimeofday(&time0, 0);
    //ctx.setheadernonce(header, sizeof(header), nonce + r);
    ctx.setheadergrin(header_data, header_length);
    printf("nonce %d k0 k1 k2 k3 %llx %llx %llx %llx\n", nonce+r,
       ctx.trimmer->sip_keys.k0, ctx.trimmer->sip_keys.k1, ctx.trimmer->sip_keys.k2, ctx.trimmer->sip_keys.k3);
    u32 nsols = ctx.solve();
    gettimeofday(&time1, 0);
    timems = (time1.tv_sec-time0.tv_sec)*1000 + (time1.tv_usec-time0.tv_usec)/1000;
    printf("Time: %d ms\n", timems);

    for (unsigned s = 0; s < nsols; s++) {
      printf("Solution");
      u32* prf = &ctx.sols[s * PROOFSIZE];
      for (u32 i = 0; i < PROOFSIZE; i++) {
        sol_nonces[i] = prf[i];
        printf(" %jx", (uintmax_t)prf[i]);
      }
      printf("\n");
      int pow_rc = verify(prf, &ctx.trimmer->sip_keys);
      if (pow_rc == POW_OK) {
        printf("Verified with cyclehash ");
        unsigned char cyclehash[32];
        blake2b((void *)cyclehash, sizeof(cyclehash), (const void *)prf, sizeof(proof), 0, 0);
        for (int i=0; i<32; i++)
          printf("%02x", cyclehash[i]);
        printf("\n");
      } else {
        printf("FAILED due to %s\n", errstr[pow_rc]);
      }
      //Just return first solution for now
      // TODO: Probably skip verify above
      if (SINGLE_MODE){
         update_stats(0, start_time);
      }
      return 1;
    }
    sumnsols += nsols;
  }
  printf("%d total solutions\n", sumnsols);
  if (SINGLE_MODE){
  printf("single mode\n");
      update_stats(0,start_time);
  }

  return 0;
}
